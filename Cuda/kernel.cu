#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <cuda/atomic>

#include "../Advent Of Code 2024 cpp/src/util.h"

#include <stdio.h>
#include <vector>
#include <string>
#include <regex>
#include <iostream>


struct Numbers {
    int64_t count;
    int64_t numbers;

    __device__ int64_t at(size_t index) {
        return (&numbers)[index];
    }
};



__device__ int64_t computeCuda(int64_t _a, int64_t _b, char _operator) {
    switch (_operator) {
    case '-':
        return _a - _b;
    case '+':
        return _a + _b;
    case '*':
        return _a * _b;
    case '/':
        return _a / _b;
    case '|':
        int multiplier = 1;
        while (multiplier <= _b) {
            multiplier *= 10;
        }
        return _a * multiplier + _b;
    }
}

__global__ void day7ComputeCuda(uint64_t* _sums, Numbers* _numbers, int64_t* _offsets, const char* _operators, int* _completeCounter) {
    int thI = threadIdx.x;
    int operatorCount = 0;
    while (_operators[operatorCount] != '\0')
    {
        operatorCount++;
    }

    int c = _numbers->count;
    _numbers = (Numbers*)(((char*)(_numbers)) + _offsets[thI]);
    c = _numbers->count;
    for (int64_t i = 0; i <= pow(operatorCount, _numbers->count - 1); i++)
    {
        int64_t result = _numbers->at(1);
        for (size_t pos = 2, j = i; pos < _numbers->count; pos++)
        {
            result = computeCuda(result, _numbers->at(pos), _operators[j % operatorCount]);
            j /= operatorCount;
        }
        if (result == _numbers->at(0))
        {
            _sums[thI] = _numbers->at(0);
            break;
        }
    }
    printf("Finished\n");
}

void day7Cuda(const std::vector<std::string>& _calibrations, std::string _operators) {
    std::vector<char> numbers;
    std::vector<int64_t> offsets = { 0 };
    int64_t lastCount = 0;
    int64_t nextFree = sizeof(int64_t);
    std::regex r("(\\d*)");

    uint64_t calibrationCount = 0;
    for (auto& i : _calibrations)
    {
        if (i.empty())
        {
            continue;
        }
        calibrationCount++;
        uint64_t count = 0;
        for (std::sregex_iterator j = std::sregex_iterator(i.begin(), i.end(), r), end = std::sregex_iterator(); j != end; j++)
        {
            if (!j->str().empty())
            {
                if (numbers.size() <= nextFree + sizeof(int64_t))
                {
                    numbers.resize(numbers.size() + 512, '\0');
                }
                *(int64_t*)(numbers.data() + nextFree) = std::stoll(j->str());
                nextFree += sizeof(int64_t);
                count++;
            }
        }
        if (numbers.size() <= lastCount + sizeof(int64_t))
        {
            numbers.resize(numbers.size() + 512, '\0');
        }
        *(int64_t*)(numbers.data() + lastCount) = count;
        lastCount = nextFree;
        offsets.emplace_back(lastCount);
        nextFree += sizeof(int64_t);
    }

    hipSetDevice(0);

    const char* operators_dev;
    uint64_t* sums_dev;
    Numbers* numbers_dev;
    int64_t* offsets_dev;

    hipMalloc(&operators_dev, _operators.size());
    hipMalloc(&sums_dev, calibrationCount * sizeof(uint64_t));
    hipMalloc(&numbers_dev, numbers.size());
    hipMalloc(&offsets_dev, offsets.size() * sizeof(uint64_t));

    hipMemcpy((void*)operators_dev, _operators.data(), _operators.size(), hipMemcpyHostToDevice);
    hipMemset(sums_dev, 0, calibrationCount * sizeof(uint64_t));
    hipMemcpy((void*)numbers_dev, numbers.data(), numbers.size(), hipMemcpyHostToDevice);
    hipMemcpy((void*)offsets_dev, offsets.data(), offsets.size() * sizeof(uint64_t), hipMemcpyHostToDevice);
    
    int completeCount = 0;
    int* completeCount_dev;
    hipMalloc(&completeCount_dev, sizeof(int));
    hipMemcpy(completeCount_dev, &completeCount, sizeof(int), hipMemcpyHostToDevice);


    day7ComputeCuda<<<1, calibrationCount>>>(sums_dev, numbers_dev, offsets_dev, operators_dev, completeCount_dev);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }

    std::vector<int64_t> sums;
    sums.resize(calibrationCount, 0);
    hipMemcpy(sums.data(), sums_dev, sums.size() * sizeof(int64_t), hipMemcpyDeviceToHost);
    
    uint64_t sum = 0;
    for (auto& i : sums)
    {
        sum += i;
    }

    hipFree((void*)operators_dev);
    hipFree(sums_dev);
    hipFree(numbers_dev);
    hipFree(offsets_dev);

    std::cout << "Day7 Part2: " << sum;

}

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
} 

int main()
{
    std::vector<std::string> calibrations = serializeInput(readInput(7));

    day7Cuda(calibrations, "-+*/|");
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
